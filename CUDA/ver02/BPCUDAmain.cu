#include "hip/hip_runtime.h"
#include "BPCUDAmain.h"
#include "Parameter.h"
#include "ReadSaveImage.h"

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

/**
* ���ܣ���ʼ�� BP �����Ȩ��
* �����weight_D Ȩ��
* ���룺row Ȩ�ص�����
* ���룺col Ȩ�ص�����
* ���룺maxNum Ȩ�ص����ֵ
*/
__global__ void Bp_Init_Weight(float *weight_D, int row, int col, float maxNum, int seed)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	hiprandState s;
	hiprand_init(index + seed, 0, 0, &s);

	if (x_id < col && y_id < row) weight_D[index] = (hiprand_uniform(&s) - 0.5f) * maxNum;
}


/**
* ���ܣ����� C = A * B'
* ���룺dev_A �����ͷָ��
* ���룺dev_B �����ͷָ��
* �����dev_C ��������ͷָ��
* ���룺heightA A ���������
* ���룺widthA A ���������
* ���룺heightB B ���������
*/
__global__ void MatMulCUDATB(float *dev_A, float *dev_B, float *dev_C, const int heightA, const int widthA, const int heightB)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	// ÿһ���̼߳���Csub�е�һ��Ԫ�أ����������Cvalue
	float Cvalue = 0;

	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	for (int m = 0; m < widthA; m += BLOCKSIZE)
	{
		int colA = m + threadIdx.x; // ��ǰ�߳��� A �е�������
		int rowB = m + threadIdx.y; // ��ǰ�߳��� B �е�������

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

		// ��Asub��Bsub�����������ڴ���
		if ((colA < widthA) && (y_id < heightA))
			As[threadIdx.y][threadIdx.x] = dev_A[y_id * widthA + colA]; // A(y_id, colA)
		else
			As[threadIdx.y][threadIdx.x] = 0.0f;

		if ((x_id < heightB) && (rowB <widthA))
			Bs[threadIdx.y][threadIdx.x] = dev_B[x_id * widthA + rowB]; // B(rowB, x_id)
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int idx = 0; idx < BLOCKSIZE; ++idx)
		{
			Cvalue += As[threadIdx.y][idx] * Bs[idx][threadIdx.x];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		__syncthreads();
	}


	if (x_id < heightB && y_id < heightA)
	{
		dev_C[y_id * heightB + x_id] = Cvalue;
	}
}

/**
* ���ܣ����������������ڻ�
* ���룺As ���� A
* ���룺Bs ���� B
* ���룺length ��������
*/
__device__ inline static float BP_Dot(float *As, float *Bs, int length)
{
	float dot = 0.0f;

	for (int i = 0; i < length; i++)
	{
		dot += As[i] * Bs[i];
	}

	return(dot);
}

__global__ void BP_Calculate_HideIn(float *dev_A, float *dev_B, float *dev_C, const int heightA, const int widthA, const int widthB)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	__shared__ float As[BLOCKSIZE_32][BLOCKSIZE_32];
	__shared__ float Bs[BLOCKSIZE_32][BLOCKSIZE_32];
	As[threadIdx.y][threadIdx.x] = 0.0f;
	Bs[threadIdx.y][threadIdx.x] = 0.0f;

	if (y_id < heightA && x_id < widthA)
	{
		As[threadIdx.y][threadIdx.x] = dev_A[threadIdx.y * widthA + x_id];
		Bs[threadIdx.y][threadIdx.x] = dev_B[threadIdx.y * widthA + x_id];
	}
	__syncthreads();

	float dot = BP_Dot(As[threadIdx.y], Bs[threadIdx.x], BLOCKSIZE_32);
	atomicAdd(&dev_C[threadIdx.y * widthB + threadIdx.x], dot);
}

/**
* ���ܣ��������ز�����
* ���룺hideOut_D ���ز�����
* �����hideOut_D ���ز����
* ���룺row Ȩ�ص�����
* ���룺col Ȩ�ص�����
*/
__global__ void BP_Calculate_HideOut(float *hideOut_D, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	if (x_id < col && y_id < row)
	{
		hideOut_D[index] = 1.0f / (1.0f + exp(-hideOut_D[index]));
	}
}

/**
* ���ܣ����� delta2_D = x_Out - A * B'
* ���룺dev_A �����ͷָ��
* ���룺dev_B �����ͷָ��
* �����delta2_D ���ز���������Ȩ������
* ���룺xOut_D �����ͷָ��
* ���룺heightA A ���������
* ���룺widthA A ���������
* ���룺heightB B ���������
*/
__global__ void BP_Calculate_Delta2(float *dev_A, float *dev_B, float *delta2_D, float *xOut_D, const int heightA, const int widthA, const int heightB)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	// ÿһ���̼߳���Csub�е�һ��Ԫ�أ����������Cvalue
	float Cvalue = 0;

	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	for (int m = 0; m < widthA; m += BLOCKSIZE)
	{
		int colA = m + threadIdx.x; // ��ǰ�߳��� A �е�������
		int rowB = m + threadIdx.y; // ��ǰ�߳��� B �е�������

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

		// ��Asub��Bsub�����������ڴ���
		if ((colA < widthA) && (y_id < heightA))
			As[threadIdx.y][threadIdx.x] = dev_A[y_id * widthA + colA]; // A(y_id, colA)
		else
			As[threadIdx.y][threadIdx.x] = 0.0f;

		if ((x_id < heightB) && (rowB <widthA))
			Bs[threadIdx.y][threadIdx.x] = dev_B[x_id * widthA + rowB]; // B(rowB, x_id)
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int idx = 0; idx < BLOCKSIZE; ++idx)
		{
			Cvalue += As[threadIdx.y][idx] * Bs[idx][threadIdx.x];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		__syncthreads();
	}


	if (x_id < heightB && y_id < heightA)
	{
		int index = y_id * heightB + x_id;
		delta2_D[index] = xOut_D[index] - Cvalue;
	}
}



/**
* ���ܣ����� C = (hOut .* (1 - hOut)) .* (A * B)
* ���룺dev_A �����ͷָ��
* ���룺dev_B �����ͷָ��
* �����dev_C ��������ͷָ��
* ���룺hideOut_D ����ͷָ��
* ���룺heightA A ���������
* ���룺widthA A ���������
* ���룺widthB B ���������
*/
__global__ void BP_Calculate_Delta1(float *dev_A, float *dev_B, float *dev_C, float *hideOut_D, const int heightA, const int widthA, const int widthB)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	// ÿһ���̼߳���Csub�е�һ��Ԫ�أ����������Cvalue
	float Cvalue = 0;

	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	for (int m = 0; m < widthA; m += BLOCKSIZE)
	{
		int colA = m + threadIdx.x; // ��ǰ�߳��� A �е�������
		int rowB = m + threadIdx.y; // ��ǰ�߳��� B �е�������

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

		// ��Asub��Bsub�����������ڴ���
		if ((colA < widthA) && (y_id < heightA))
			As[threadIdx.y][threadIdx.x] = dev_A[y_id * widthA + colA]; // A(y_id, colA)
		else
			As[threadIdx.y][threadIdx.x] = 0.0f;

		if ((x_id < widthB) && (rowB <widthA))
			Bs[threadIdx.y][threadIdx.x] = dev_B[rowB * widthB + x_id]; // B(rowB, x_id)
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int idx = 0; idx < BLOCKSIZE; ++idx)
		{
			Cvalue += As[threadIdx.y][idx] * Bs[idx][threadIdx.x];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		__syncthreads();
	}

	if (x_id < widthB && y_id < heightA)
	{
		int index = y_id * widthB + x_id;
		float data = hideOut_D[index];
		dev_C[index] = data * (1.0f - data) * Cvalue;
	}
}

/**
* ���ܣ�����Ȩ�� C = C + eta/batchNum .* (A' * B)
* ���룺dev_A �����ͷָ��
* ���룺dev_B �����ͷָ��
* �����dev_C ��������ͷָ��
* ���룺heightA A ���������
* ���룺widthA A ���������
* ���룺heightB B ���������
*/
__global__ void BP_Update_Weight(float *dev_A, float *dev_B, float *dev_C, const int heightA, const int widthA, const int widthB)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	// ÿһ���̼߳���Csub�е�һ��Ԫ�أ����������Cvalue
	float Cvalue = 0;

	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	for (int m = 0; m < heightA; m += BLOCKSIZE)
	{
		int colA = m + threadIdx.x; // ��ǰ�߳��� A �е�������
		int rowB = m + threadIdx.y; // ��ǰ�߳��� B �е�������

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

		// ��Asub��Bsub�����������ڴ���
		if ((colA < heightA) && (y_id < widthA))
			As[threadIdx.y][threadIdx.x] = dev_A[colA * widthA + y_id]; // A(y_id, colA)
		else
			As[threadIdx.y][threadIdx.x] = 0.0f;

		if ((x_id < widthB) && (rowB < heightA))
			Bs[threadIdx.y][threadIdx.x] = dev_B[rowB * widthB + x_id]; // B(rowB, x_id)
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int idx = 0; idx < BLOCKSIZE; ++idx)
		{
			Cvalue += As[threadIdx.y][idx] * Bs[idx][threadIdx.x];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		__syncthreads();
	}

	if (x_id < widthB && y_id < widthA)
	{
		dev_C[y_id * widthB + x_id] += eta  * Cvalue / float(batchNum);
	}
}

/**
* ���ܣ�������������ݸ������ǩ
* �����yOutTestClass_D ÿ���������������
* ���룺yOutTest_D ÿ��������Ӧ�����
* ���룺row ������
* ���룺col ���������˴�Ϊ 10
*/
__global__ void BP_Calculate_Class(int *yOutTestClass_D, float *yOutTest_D, int row, int col)
{
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	__shared__ float sData[BLOCKSIZE][BLOCKSIZE]; // ÿ�����������
	__shared__ int sIndx[BLOCKSIZE][BLOCKSIZE]; // �����Ӧ������

	if (threadIdx.x < BLOCKSIZE / 2)
	{
		sData[threadIdx.y][threadIdx.x] = 0;
		sIndx[threadIdx.y][threadIdx.x] = threadIdx.x;
		sData[threadIdx.y][threadIdx.x + BLOCKSIZE / 2] = -2e30;
		sIndx[threadIdx.y][threadIdx.x + BLOCKSIZE / 2] = threadIdx.x + BLOCKSIZE / 2;
	}

	__syncthreads();

	if (y_id < row && threadIdx.x < col)
	{
		float *objIndex = &yOutTest_D[y_id * col];
		sData[threadIdx.y][threadIdx.x] = objIndex[threadIdx.x];

		__syncthreads();

		/* BLOCKSIZE �����ڲ���Լ����ֻʣ 2 �� */
		for (int step = BLOCKSIZE / 2; step > 1; step = step >> 1)
		{
			int idxStep = threadIdx.x + step;
			if (threadIdx.x < step && sData[threadIdx.y][threadIdx.x] < sData[threadIdx.y][idxStep])
			{
				sData[threadIdx.y][threadIdx.x] = sData[threadIdx.y][idxStep];
				sIndx[threadIdx.y][threadIdx.x] = sIndx[threadIdx.y][idxStep];
			}
		}

		if (threadIdx.x == 0)
		{
			yOutTestClass_D[y_id] = sData[threadIdx.y][0] > sData[threadIdx.y][1] ? sIndx[threadIdx.y][0] : sIndx[threadIdx.y][1];
		}
	}
}

/**
* ���ܣ�������������ݸ������ǩ
* �����yOutTestClass_D ÿ���������������
* ���룺yOutTest_D ÿ��������Ӧ�����
* ���룺row ������
* ���룺col ���������˴�Ϊ 10
*/
__global__ void BP_Calculate_RightRidio(int *yOutTestClass_D, int *outputTestClass_D, int row, int *wrongNum)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������

	if (x_id < row && yOutTestClass_D[x_id] != outputTestClass_D[x_id])
	{
		//printf("x_id = %d, real = %d, test = %d\n", x_id, outputTestClass_D[x_id], yOutTestClass_D[x_id]);
		atomicAdd((int*)&wrongNum[0], 1);
	}
}

/*
* ���ܣ�BP �㷨ʵ����������д����ʶ��
* ���룺inputTrain_H �����ѵ������
* ���룺inputTest_H ����Ĳ�������
* ���룺outputTrain_H ѵ���������������ǩ��
* ���룺outputTest_H  �����������������ǩ��
*/
void BpMain(float *inputTrain_H, float *inputTest_H, float *outputTrain_H, float *outputTest_H)
{
	/* �����豸���ڴ� */
	float *inputTrain_D, *inputTest_D, *outputTrain_D, *outputTest_D;
	hipMalloc((void**)&inputTrain_D, trainNum * inLayout * sizeof(float));
	hipMalloc((void**)&inputTest_D, testNum * inLayout * sizeof(float));
	hipMalloc((void**)&outputTrain_D, trainNum * outLayout * sizeof(float));
	hipMalloc((void**)&outputTest_D, testNum * outLayout * sizeof(float));

	float *weightHideIn_D, *weightOutHide_D;
	hipMalloc((void**)&weightHideIn_D, hideLayout * inLayout * sizeof(float));
	hipMalloc((void**)&weightOutHide_D, outLayout * hideLayout * sizeof(float));

	float *weightHideInT_D;
	hipMalloc((void**)&weightHideInT_D, hideLayout * inLayout * sizeof(float));

	float *deltaHideIn_D, *deltaOutHide_D;
	hipMalloc((void**)&deltaHideIn_D, hideLayout * batchNum * sizeof(float));
	hipMalloc((void**)&deltaOutHide_D, outLayout * batchNum * sizeof(float));

	float *hideOut_D, *hideOutTest_D;
	hipMalloc((void**)&hideOut_D, hideLayout * batchNum * sizeof(float));
	hipMemset(hideOut_D, 0, hideLayout * batchNum * sizeof(float));
	hipMalloc((void**)&hideOutTest_D, hideLayout * testNum * sizeof(float));

	float *phi_D;
	hipMalloc((void**)&phi_D, hideLayout * batchNum * sizeof(float));

	float *yOut_D, *yOutTest_D;
	hipMalloc((void**)&yOut_D, outLayout * batchNum * sizeof(float));
	hipMalloc((void**)&yOutTest_D, outLayout * testNum * sizeof(float));

	int *yOutTestClass_D, *outputTestClass_D;
	hipMalloc((void**)&yOutTestClass_D, testNum * sizeof(int));
	hipMalloc((void**)&outputTestClass_D, testNum * sizeof(int));

	float *w10 = (float*)malloc(hideLayout * inLayout * sizeof(float));
	float *w21 = (float*)malloc(outLayout * hideLayout * sizeof(float));

	int *wrongNum_H = (int*)malloc(sizeof(int));
	int *wrongNum_D;
	hipMalloc((void**)&wrongNum_D, sizeof(int));
	hipMemset(wrongNum_D, 0, sizeof(int));

	/* ���ݴ������˿������豸�� */
	hipMemcpy(inputTrain_D, inputTrain_H, trainNum * inLayout * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(inputTest_D, inputTest_H, testNum * inLayout * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(outputTrain_D, outputTrain_H, trainNum * outLayout * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(outputTest_D, outputTest_H, testNum * outLayout * sizeof(float), hipMemcpyHostToDevice);


	//std::string strW10 = "D:\\Document\\vidpic\\CUDA\\BP\\data\\6000\\W10.txt";
	//std::string strW21 = "D:\\Document\\vidpic\\CUDA\\BP\\data\\6000\\W21.txt";

	//ReadFile(w10, strW10, hideLayout * inLayout);
	//ReadFile(w21, strW21, outLayout * hideLayout);

	//hipMemcpy(weightHideIn_D, w10, hideLayout * inLayout * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(weightOutHide_D, w21, outLayout * hideLayout * sizeof(float), hipMemcpyHostToDevice);

	/* �����̸߳���߳̿� */
	dim3 dimBlock2D(BLOCKSIZE, BLOCKSIZE);
	dim3 dimBlock2D_32(BLOCKSIZE_32, BLOCKSIZE_32);
	dim3 dimBlock1D(BLOCKSIZE * BLOCKSIZE);
	dim3 dimGrid2D_hide_in((inLayout + BLOCKSIZE - 1) / dimBlock2D.x, (hideLayout + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid2D_out_hide((hideLayout + BLOCKSIZE - 1) / dimBlock2D.x, (outLayout + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid2D_batch_hide((hideLayout + BLOCKSIZE - 1) / dimBlock2D.x, (batchNum + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid2D_batch_out((outLayout + BLOCKSIZE - 1) / dimBlock2D.x, (batchNum + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid2D_testNum_hide((hideLayout + BLOCKSIZE - 1) / dimBlock2D.x, (testNum + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid2D_testNum_out((outLayout + BLOCKSIZE - 1) / dimBlock2D.x, (testNum + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid1D_testNum(((testNum + BLOCKSIZE - 1) / dimBlock2D.x));
	dim3 dimGrid2D_32_batch_in((inLayout + BLOCKSIZE_32 - 1) / dimBlock2D_32.x, (batchNum + BLOCKSIZE_32 - 1) / dimBlock2D_32.y);

	/* ��¼ʱ�� */
	hipEvent_t start_GPU, end_GPU;
	float elaspsedTime;
	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	/* Ȩ�س�ʼ�� */
	Bp_Init_Weight<<<dimGrid2D_hide_in, dimBlock2D>>>(weightHideIn_D, hideLayout, inLayout, initWeightMax, 0);
	Bp_Init_Weight<<<dimGrid2D_out_hide, dimBlock2D>>>(weightOutHide_D, outLayout, hideLayout, initWeightMax, 393);

	for (int i = 0; i < 50; i++)
	{
		for (int batch = 0; batch < trainNum; batch += batchNum)
		{
			/* hIn = X * W01' */
			BP_Calculate_HideIn<<<dimGrid2D_32_batch_in, dimBlock2D_32>>>(&inputTrain_D[batch * inLayout], weightHideIn_D, hideOut_D, batchNum, inLayout, hideLayout);

			/* hOut = h(hIn) */
			BP_Calculate_HideOut<<<dimGrid2D_batch_hide, dimBlock2D>>>(hideOut_D, batchNum, hideLayout);

			/* delta2 = xOut - hOut * W21' */
			BP_Calculate_Delta2<<<dimGrid2D_batch_out, dimBlock2D>>>(hideOut_D, weightOutHide_D, deltaOutHide_D, &outputTrain_D[batch * outLayout], batchNum, hideLayout, outLayout);

			/* delta1 = (hOut .* (1 - hOut)) .* (delta2 * W21) */
			BP_Calculate_Delta1<<<dimGrid2D_batch_hide, dimBlock2D>>>(deltaOutHide_D, weightOutHide_D, deltaHideIn_D, hideOut_D, batchNum, outLayout, hideLayout);

			/* W21 = W21 + eta / batchNum * delta2' * hOut */
			BP_Update_Weight<<<dimGrid2D_out_hide, dimBlock2D>>>(deltaOutHide_D, hideOut_D, weightOutHide_D, batchNum, outLayout, hideLayout);

			/* W10 = W10 + eta / batchNum * delta1' * X */
			BP_Update_Weight<<<dimGrid2D_hide_in, dimBlock2D>>>(deltaHideIn_D, &inputTrain_D[batch * inLayout], weightHideIn_D, batchNum, hideLayout, inLayout);
		}
	}

	/* ������� */
	/* hIn = X * W01' */
	MatMulCUDATB<<<dimGrid2D_testNum_hide, dimBlock2D>>>(inputTest_D, weightHideIn_D, hideOutTest_D, testNum, inLayout, hideLayout);

	/* hOut = h(hIn) */
	BP_Calculate_HideOut<<<dimGrid2D_testNum_hide, dimBlock2D>>>(hideOutTest_D, testNum, hideLayout);

	/* yOut = hOut * W21' */
	MatMulCUDATB<<<dimGrid2D_testNum_out, dimBlock2D>>>(hideOutTest_D, weightOutHide_D, yOutTest_D, testNum, hideLayout, outLayout);

	/* [output_result, ~] = find(bsxfun(@eq, yOut, max(yOut)) ~= 0); */
	BP_Calculate_Class<<<dimGrid2D_testNum_out, dimBlock2D>>>(yOutTestClass_D, yOutTest_D, testNum, outLayout);
	BP_Calculate_Class<<<dimGrid2D_testNum_out, dimBlock2D>>>(outputTestClass_D, outputTest_D, testNum, outLayout);
	
	/* ����׼ȷ�� */
	BP_Calculate_RightRidio<<<dimGrid1D_testNum, dimBlock1D>>>(yOutTestClass_D, outputTestClass_D, testNum, wrongNum_D);

	/* ��ʱ���� */
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	/* ��ӡ��Ϣ */
	std::cout << "BP ��ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	hipMemcpy(wrongNum_H, wrongNum_D, sizeof(int), hipMemcpyDeviceToHost);
	printf("BP �ľ���Ϊ��%.2f%%\n", 100.0f*float(testNum - *wrongNum_H) / float(testNum));

	hipMemcpy(w10, weightHideIn_D, hideLayout * inLayout * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(w21, weightOutHide_D, outLayout * hideLayout * sizeof(float), hipMemcpyDeviceToHost);

	std::string strW10result = "D:\\Document\\vidpic\\CUDA\\BP\\data\\6000\\W10result.txt";
	std::string strW21result = "D:\\Document\\vidpic\\CUDA\\BP\\data\\6000\\W21result.txt";

	SaveFile(w10, strW10result, hideLayout * inLayout);
	SaveFile(w21, strW21result, outLayout * hideLayout);

	/* �ͷ��豸���ڴ� */
	hipFree(inputTrain_D);
	hipFree(inputTest_D);
	hipFree(outputTrain_D);
	hipFree(outputTest_D);
}
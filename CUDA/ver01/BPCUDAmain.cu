#include "hip/hip_runtime.h"
#include "BPCUDAmain.h"
#include "Parameter.h"
#include "ReadSaveImage.h"

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

/**
* ���ܣ���ʼ�� BP �����Ȩ��
* �����weight_D Ȩ��
* ���룺row Ȩ�ص�����
* ���룺col Ȩ�ص�����
* ���룺maxNum Ȩ�ص����ֵ
*/
__global__ void Bp_Init_Weight(float *weight_D, int row, int col, float maxNum, int seed)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	hiprandState s;
	hiprand_init(index + seed, 0, 0, &s);

	if (x_id < col && y_id < row) weight_D[index] = (hiprand_uniform(&s) - 0.5f) * maxNum;
}

/**
* ���ܣ����� C = A * B
* ���룺dev_A �����ͷָ��
* ���룺dev_B �����ͷָ��
* �����dev_C ��������ͷָ��
* ���룺heightA A ���������
* ���룺widthA A ���������
* ���룺widthB B ���������
*/
__global__ void MatMulCUDA(float *dev_A, float *dev_B, float *dev_C, const int heightA, const int widthA, const int widthB)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	// ÿһ���̼߳���Csub�е�һ��Ԫ�أ����������Cvalue
	float Cvalue = 0;

	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	for (int m = 0; m < widthA; m += BLOCKSIZE)
	{
		int colA = m + threadIdx.x; // ��ǰ�߳��� A �е�������
		int rowB = m + threadIdx.y; // ��ǰ�߳��� B �е�������

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

		// ��Asub��Bsub�����������ڴ���
		if ((colA < widthA) && (y_id < heightA))
			As[threadIdx.y][threadIdx.x] = dev_A[y_id * widthA + colA]; // A(y_id, colA)
		else
			As[threadIdx.y][threadIdx.x] = 0.0f;

		if ((x_id < widthB) && (rowB <widthA))
			Bs[threadIdx.y][threadIdx.x] = dev_B[rowB * widthB + x_id]; // B(rowB, x_id)
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int idx = 0; idx < BLOCKSIZE; ++idx)
		{
			Cvalue += As[threadIdx.y][idx] * Bs[idx][threadIdx.x];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		__syncthreads();
	}

	if (x_id < widthB && y_id < heightA)
	{
		dev_C[y_id * widthB + x_id] = Cvalue;
	}
}


/**
* ���ܣ����� C = A * B'
* ���룺dev_A �����ͷָ��
* ���룺dev_B �����ͷָ��
* �����dev_C ��������ͷָ��
* ���룺heightA A ���������
* ���룺widthA A ���������
* ���룺heightB B ���������
*/
__global__ void MatMulCUDATB(float *dev_A, float *dev_B, float *dev_C, const int heightA, const int widthA, const int heightB, int flag)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	// ÿһ���̼߳���Csub�е�һ��Ԫ�أ����������Cvalue
	float Cvalue = 0;

	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	for (int m = 0; m < widthA; m += BLOCKSIZE)
	{
		int colA = m + threadIdx.x; // ��ǰ�߳��� A �е�������
		int rowB = m + threadIdx.y; // ��ǰ�߳��� B �е�������

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

		// ��Asub��Bsub�����������ڴ���
		if ((colA < widthA) && (y_id < heightA))
			As[threadIdx.y][threadIdx.x] = dev_A[y_id * widthA + colA]; // A(y_id, colA)
		else
			As[threadIdx.y][threadIdx.x] = 0.0f;

		if ((x_id < heightB) && (rowB <widthA))
			Bs[threadIdx.y][threadIdx.x] = dev_B[x_id * widthA + rowB]; // B(rowB, x_id)
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int idx = 0; idx < BLOCKSIZE; ++idx)
		{
			Cvalue += As[threadIdx.y][idx] * Bs[idx][threadIdx.x];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		__syncthreads();
	}


	if (x_id < heightB && y_id < heightA)
	{
		if (flag == 1 && x_id < 32 && y_id >= 800 && y_id < 840) printf("x_id = %d, y_id = %d, C = %f\n", x_id, y_id, Cvalue);
		dev_C[y_id * heightB + x_id] = Cvalue;
	}
}




/**
* ���ܣ��������ز�����
* ���룺hideOut_D ���ز�����
* �����hideOut_D ���ز����
* ���룺row Ȩ�ص�����
* ���룺col Ȩ�ص�����
*/
__global__ void BP_Calculate_Hide(float *hideOut_D, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	if (x_id < col && y_id < row)
	{
		hideOut_D[index] = 1.0f / (1.0f + exp(-hideOut_D[index]));
	}
}

/**
* ���ܣ����� phi
* ���룺hideOut_D ���ز�����
* �����hideOut_D ���ز����
* ���룺row Ȩ�ص�����
* ���룺col Ȩ�ص�����
*/
__global__ void BP_Calculate_Phi(float *hideOut_D, float *phi_D, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	if (x_id < col && y_id < row)
	{
		float data = hideOut_D[index];
		phi_D[index] = data * (1.0f - data);
	}
}

/**
* ���ܣ����� delta2 = xOut - yOut
* �����delta2_D ���ز���������Ȩ������
* ���룺xOut_D �������
* ���룺yOut_D ʵ�����
* ���룺row Ȩ�ص�����
* ���룺col Ȩ�ص�����
*/
__global__ void BP_Calculate_Delta2(float *delta2_D, float *xOut_D, float *yOut_D, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	if (x_id < col && y_id < row)
	{
		delta2_D[index] = xOut_D[index] - yOut_D[index];
	}
}

/**
* ���ܣ����� delta1 = phi .* (W21' * yOut)
* ���룺hideOut_D ���ز�����
* �����hideOut_D ���ز����
* ���룺row Ȩ�ص�����
* ���룺col Ȩ�ص�����
*/
__global__ void BP_Calculate_Delta1(float *delta1_D, float *phi_D, int row, int col)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������
	int index = y_id * col + x_id;

	if (x_id < col && y_id < row)
	{
		delta1_D[index] *= phi_D[index];
	}
}


/**
* ���ܣ�����Ȩ�� C = C + eta/batchNum .* (A' * B)
* ���룺dev_A �����ͷָ��
* ���룺dev_B �����ͷָ��
* �����dev_C ��������ͷָ��
* ���룺heightA A ���������
* ���룺widthA A ���������
* ���룺heightB B ���������
*/
__global__ void BP_Update_Weight(float *dev_A, float *dev_B, float *dev_C, const int heightA, const int widthA, const int widthB, int flag)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	// ÿһ���̼߳���Csub�е�һ��Ԫ�أ����������Cvalue
	float Cvalue = 0;

	// A�����ӿ� * B�����ӿ� = ��ӦC���ӿ�Csub
	for (int m = 0; m < heightA; m += BLOCKSIZE)
	{
		int colA = m + threadIdx.x; // ��ǰ�߳��� A �е�������
		int rowB = m + threadIdx.y; // ��ǰ�߳��� B �е�������

		// ���乲���ڴ�ռ䣬�������Asub��Bsub
		__shared__ float As[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Bs[BLOCKSIZE][BLOCKSIZE];

		// ��Asub��Bsub�����������ڴ���
		if ((colA < heightA) && (y_id < widthA))
			As[threadIdx.y][threadIdx.x] = dev_A[colA * widthA + y_id]; // A(y_id, colA)
		else
			As[threadIdx.y][threadIdx.x] = 0.0f;

		if ((x_id < widthB) && (rowB < heightA))
			Bs[threadIdx.y][threadIdx.x] = dev_B[rowB * widthB + x_id]; // B(rowB, x_id)
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0f;

		__syncthreads();

		// A�ӿ����*B�ӿ����
		// �ӿ��ڵ�ѭ��
		for (int idx = 0; idx < BLOCKSIZE; ++idx)
		{
			Cvalue += As[threadIdx.y][idx] * Bs[idx][threadIdx.x];
		}

		// ͬ��,ȷ����ǰA�ӿ���B�ӿ�ļ������
		__syncthreads();
	}

	if (x_id < widthB && y_id < widthA)
	{
		dev_C[y_id * widthB + x_id] += eta  * Cvalue / float(batchNum);
	}
}

/**
* ���ܣ�������������ݸ������ǩ
* �����yOutTestClass_D ÿ���������������
* ���룺yOutTest_D ÿ��������Ӧ�����
* ���룺row ������
* ���룺col ���������˴�Ϊ 10
*/
__global__ void BP_Calculate_Class(int *yOutTestClass_D, float *yOutTest_D, int row, int col)
{
	int y_id = blockDim.y * blockIdx.y + threadIdx.y; // ������

	__shared__ float sData[BLOCKSIZE][BLOCKSIZE]; // ÿ�����������
	__shared__ int sIndx[BLOCKSIZE][BLOCKSIZE]; // �����Ӧ������

	if (threadIdx.x < BLOCKSIZE / 2)
	{
		sData[threadIdx.y][threadIdx.x] = 0;
		sIndx[threadIdx.y][threadIdx.x] = threadIdx.x;
		sData[threadIdx.y][threadIdx.x + BLOCKSIZE / 2] = -2e30;
		sIndx[threadIdx.y][threadIdx.x + BLOCKSIZE / 2] = threadIdx.x + BLOCKSIZE / 2;
	}

	__syncthreads();

	if (y_id < row && threadIdx.x < col)
	{
		float *objIndex = &yOutTest_D[y_id * col];
		sData[threadIdx.y][threadIdx.x] = objIndex[threadIdx.x];

		__syncthreads();

		/* BLOCKSIZE �����ڲ���Լ����ֻʣ 2 �� */
		for (int step = BLOCKSIZE / 2; step > 1; step = step >> 1)
		{
			int idxStep = threadIdx.x + step;
			if (threadIdx.x < step && sData[threadIdx.y][threadIdx.x] < sData[threadIdx.y][idxStep])
			{
				sData[threadIdx.y][threadIdx.x] = sData[threadIdx.y][idxStep];
				sIndx[threadIdx.y][threadIdx.x] = sIndx[threadIdx.y][idxStep];
			}
		}

		if (threadIdx.x == 0)
		{
			yOutTestClass_D[y_id] = sData[threadIdx.y][0] > sData[threadIdx.y][1] ? sIndx[threadIdx.y][0] : sIndx[threadIdx.y][1];
		}
	}
}

/**
* ���ܣ�������������ݸ������ǩ
* �����yOutTestClass_D ÿ���������������
* ���룺yOutTest_D ÿ��������Ӧ�����
* ���룺row ������
* ���룺col ���������˴�Ϊ 10
*/
__global__ void BP_Calculate_RightRidio(int *yOutTestClass_D, int *outputTestClass_D, int row, int *wrongNum)
{
	int x_id = blockDim.x * blockIdx.x + threadIdx.x; // ������

	if (x_id < row && yOutTestClass_D[x_id] != outputTestClass_D[x_id])
	{
		//printf("x_id = %d, real = %d, test = %d\n", x_id, outputTestClass_D[x_id], yOutTestClass_D[x_id]);
		atomicAdd((int*)&wrongNum[0], 1);
	}
}

/*
* ���ܣ�BP �㷨ʵ����������д����ʶ��
* ���룺inputTrain_H �����ѵ������
* ���룺inputTest_H ����Ĳ�������
* ���룺outputTrain_H ѵ���������������ǩ��
* ���룺outputTest_H  �����������������ǩ��
*/
void BpMain(float *inputTrain_H, float *inputTest_H, float *outputTrain_H, float *outputTest_H)
{
	/* �����豸���ڴ� */
	float *inputTrain_D, *inputTest_D, *outputTrain_D, *outputTest_D;
	hipMalloc((void**)&inputTrain_D, trainNum * inLayout * sizeof(float));
	hipMalloc((void**)&inputTest_D, testNum * inLayout * sizeof(float));
	hipMalloc((void**)&outputTrain_D, trainNum * outLayout * sizeof(float));
	hipMalloc((void**)&outputTest_D, testNum * outLayout * sizeof(float));

	float *weightHideIn_D, *weightOutHide_D;
	hipMalloc((void**)&weightHideIn_D, hideLayout * inLayout * sizeof(float));
	hipMalloc((void**)&weightOutHide_D, outLayout * hideLayout * sizeof(float));

	float *deltaHideIn_D, *deltaOutHide_D;
	hipMalloc((void**)&deltaHideIn_D, hideLayout * batchNum * sizeof(float));
	hipMalloc((void**)&deltaOutHide_D, outLayout * batchNum * sizeof(float));

	float *hideOut_D, *hideOutTest_D;
	hipMalloc((void**)&hideOut_D, hideLayout * batchNum * sizeof(float));
	hipMalloc((void**)&hideOutTest_D, hideLayout * testNum * sizeof(float));

	float *phi_D;
	hipMalloc((void**)&phi_D, hideLayout * batchNum * sizeof(float));

	float *yOut_D, *yOutTest_D;
	hipMalloc((void**)&yOut_D, outLayout * batchNum * sizeof(float));
	hipMalloc((void**)&yOutTest_D, outLayout * testNum * sizeof(float));

	int *yOutTestClass_D, *outputTestClass_D;
	hipMalloc((void**)&yOutTestClass_D, testNum * sizeof(int));
	hipMalloc((void**)&outputTestClass_D, testNum * sizeof(int));

	float *w10 = (float*)malloc(hideLayout * inLayout * sizeof(float));
	float *w21 = (float*)malloc(outLayout * hideLayout * sizeof(float));

	/* ���ݴ������˿������豸�� */
	hipMemcpy(inputTrain_D, inputTrain_H, trainNum * inLayout * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(inputTest_D, inputTest_H, testNum * inLayout * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(outputTrain_D, outputTrain_H, trainNum * outLayout * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(outputTest_D, outputTest_H, testNum * outLayout * sizeof(float), hipMemcpyHostToDevice);


	//std::string strW10 = "D:\\Document\\vidpic\\CUDA\\BP\\data\\6000\\W10.txt";
	//std::string strW21 = "D:\\Document\\vidpic\\CUDA\\BP\\data\\6000\\W21.txt";

	//ReadFile(w10, strW10, hideLayout * inLayout);
	//ReadFile(w21, strW21, outLayout * hideLayout);

	//hipMemcpy(weightHideIn_D, w10, hideLayout * inLayout * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(weightOutHide_D, w21, outLayout * hideLayout * sizeof(float), hipMemcpyHostToDevice);

	/* �����̸߳���߳̿� */
	dim3 dimBlock2D(BLOCKSIZE, BLOCKSIZE);
	dim3 dimBlock1D(BLOCKSIZE * BLOCKSIZE);
	dim3 dimGrid2D_hide_in((inLayout + BLOCKSIZE - 1) / dimBlock2D.x, (hideLayout + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid2D_out_hide((hideLayout + BLOCKSIZE - 1) / dimBlock2D.x, (outLayout + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid2D_batch_hide((hideLayout + BLOCKSIZE - 1) / dimBlock2D.x, (batchNum + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid2D_batch_out((outLayout + BLOCKSIZE - 1) / dimBlock2D.x, (batchNum + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid2D_testNum_hide((hideLayout + BLOCKSIZE - 1) / dimBlock2D.x, (testNum + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid2D_testNum_out((outLayout + BLOCKSIZE - 1) / dimBlock2D.x, (testNum + BLOCKSIZE - 1) / dimBlock2D.y);
	dim3 dimGrid1D_testNum(((testNum + BLOCKSIZE - 1) / dimBlock2D.x));

	/* ��¼ʱ�� */
	hipEvent_t start_GPU, end_GPU;
	float elaspsedTime;
	hipEventCreate(&start_GPU);
	hipEventCreate(&end_GPU);
	hipEventRecord(start_GPU, 0);

	/* Ȩ�س�ʼ�� */
	Bp_Init_Weight<<<dimGrid2D_hide_in, dimBlock2D>>>(weightHideIn_D, hideLayout, inLayout, initWeightMax, 0);
	Bp_Init_Weight<<<dimGrid2D_out_hide, dimBlock2D>>>(weightOutHide_D, outLayout, hideLayout, initWeightMax, 393);

	for (int i = 0; i < 50; i++)
	{
		for (int batch = 0; batch < trainNum; batch += batchNum)
		{
			/* hIn = X * W01' */
			MatMulCUDATB<<<dimGrid2D_batch_hide, dimBlock2D>>>(&inputTrain_D[batch * inLayout], weightHideIn_D, hideOut_D, batchNum, inLayout, hideLayout, 0);

			/* hOut = h(hIn) */
			BP_Calculate_Hide<<<dimGrid2D_batch_hide, dimBlock2D>>>(hideOut_D, batchNum, hideLayout);

			/* yOut = hOut * W21' */
			MatMulCUDATB<<<dimGrid2D_batch_out, dimBlock2D>>>(hideOut_D, weightOutHide_D, yOut_D, batchNum, hideLayout, outLayout, 0);

			/* delta2 = xOut - yOut */
			BP_Calculate_Delta2<<<dimGrid2D_batch_out, dimBlock2D>>>(deltaOutHide_D, &outputTrain_D[batch * outLayout], yOut_D, batchNum, outLayout);

			/* phi = hOut .* (1 - hOut) */
			BP_Calculate_Phi<<<dimGrid2D_batch_hide, dimBlock2D>>>(hideOut_D, phi_D, batchNum, hideLayout);

			/* delta1 = phi .* (delta2 * W21) */
			MatMulCUDA<<<dimGrid2D_batch_hide, dimBlock2D>>>(deltaOutHide_D, weightOutHide_D, deltaHideIn_D, batchNum, outLayout, hideLayout);
			BP_Calculate_Delta1<<<dimGrid2D_batch_hide, dimBlock2D>>>(deltaHideIn_D, phi_D, batchNum, hideLayout);

			/* W21 = W21 + eta / batchNum * delta2' * hOut */
			BP_Update_Weight<<<dimGrid2D_out_hide, dimBlock2D>>>(deltaOutHide_D, hideOut_D, weightOutHide_D, batchNum, outLayout, hideLayout, 0);

			/* W10 = W10 + eta / batchNum * delta1' * X */
			BP_Update_Weight<<<dimGrid2D_hide_in, dimBlock2D>>>(deltaHideIn_D, &inputTrain_D[batch * inLayout], weightHideIn_D, batchNum, hideLayout, inLayout, 0);
		}
	}

	/* ������� */
	/* hIn = X * W01' */
	MatMulCUDATB<<<dimGrid2D_testNum_hide, dimBlock2D>>>(inputTest_D, weightHideIn_D, hideOutTest_D, testNum, inLayout, hideLayout, 0);

	/* hOut = h(hIn) */
	BP_Calculate_Hide<<<dimGrid2D_testNum_hide, dimBlock2D>>>(hideOutTest_D, testNum, hideLayout);

	/* yOut = hOut * W21' */
	MatMulCUDATB<<<dimGrid2D_testNum_out, dimBlock2D>>>(hideOutTest_D, weightOutHide_D, yOutTest_D, testNum, hideLayout, outLayout, 0);

	/* [output_result, ~] = find(bsxfun(@eq, yOut, max(yOut)) ~= 0); */
	BP_Calculate_Class<<<dimGrid2D_testNum_out, dimBlock2D>>>(yOutTestClass_D, yOutTest_D, testNum, outLayout);
	BP_Calculate_Class<<<dimGrid2D_testNum_out, dimBlock2D>>>(outputTestClass_D, outputTest_D, testNum, outLayout);
	
	/* */
	int *wrongNum_H = (int*)malloc(sizeof(int));
	int *wrongNum_D;
	hipMalloc((void**)&wrongNum_D, sizeof(int));
	hipMemset(wrongNum_D, 0, sizeof(int));

	BP_Calculate_RightRidio<<<dimGrid1D_testNum, dimBlock1D>>>(yOutTestClass_D, outputTestClass_D, testNum, wrongNum_D);

	hipMemcpy(wrongNum_H, wrongNum_D, sizeof(int), hipMemcpyDeviceToHost);
	printf("׼ȷ��Ϊ�� %f\%\n", 100.0f*float(testNum - *wrongNum_H) / float(testNum));

	/* ��ʱ���� */
	hipEventRecord(end_GPU, 0);
	hipEventSynchronize(end_GPU);
	hipEventElapsedTime(&elaspsedTime, start_GPU, end_GPU);

	/* ��ӡ��Ϣ */
	std::cout << "BP ��ʱ��Ϊ��" << elaspsedTime << "ms." << std::endl;

	hipMemcpy(w10, weightHideIn_D, hideLayout * inLayout * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(w21, weightOutHide_D, outLayout * hideLayout * sizeof(float), hipMemcpyDeviceToHost);

	std::string strW10result = "D:\\Document\\vidpic\\CUDA\\BP\\data\\6000\\W10result.txt";
	std::string strW21result = "D:\\Document\\vidpic\\CUDA\\BP\\data\\6000\\W21result.txt";

	SaveFile(w10, strW10result, hideLayout * inLayout);
	SaveFile(w21, strW21result, outLayout * hideLayout);

	/* �ͷ��豸���ڴ� */
	hipFree(inputTrain_D);
	hipFree(inputTest_D);
	hipFree(outputTrain_D);
	hipFree(outputTest_D);
}